#include "hip/hip_runtime.h"
#include <cstdio>
#include "../../structures/Ray.h"

float* d_x;
float* d_y;
float* d_z;
int num_vertices;

__global__ void linear_intersect_kernel(Polytope::Ray &ray) {
   
}

void linear_intersect(Polytope::Ray &ray) {
   const int threadsPerBlock = 256;
   const int blocksPerGrid = (num_vertices + threadsPerBlock - 1) / threadsPerBlock;
   linear_intersect_kernel<<<blocksPerGrid, threadsPerBlock>>>(ray);
}

void initialize_unpacked_mesh(const float *h_x, const float *h_y, const float *h_z, const int num_verts) {
   num_vertices = num_verts;
   
   hipError_t err;
   size_t size = num_verts * sizeof(float);
   
   err = hipMalloc((void **)&d_x, size);

   if (err != hipSuccess)
   {
      fprintf(stderr, "Failed to allocate device vector x (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }

   err = hipMalloc((void **)&d_y, size);

   if (err != hipSuccess)
   {
      fprintf(stderr, "Failed to allocate device vector y (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }

   err = hipMalloc((void **)&d_z, size);

   if (err != hipSuccess)
   {
      fprintf(stderr, "Failed to allocate device vector z (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }

   err = hipMemcpy(d_x, h_x, size, ::hipMemcpyHostToDevice);

   if (err != hipSuccess)
   {
      fprintf(stderr, "Failed to copy vector x from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }

   err = hipMemcpy(d_y, h_y, size, ::hipMemcpyHostToDevice);

   if (err != hipSuccess)
   {
      fprintf(stderr, "Failed to copy vector y from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }

   err = hipMemcpy(d_z, h_z, size, ::hipMemcpyHostToDevice);

   if (err != hipSuccess)
   {
      fprintf(stderr, "Failed to copy vector z from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }
}



void free_mesh() {
   hipError_t err;
   err = hipFree(d_x);
   err = hipFree(d_y);
   err = hipFree(d_z);
}